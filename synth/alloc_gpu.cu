#ifndef ALLOC_GPU_H
#define ALLOC_GPU_H

#include "gpu_assert.cu"

void* alloc(size_t size) {
    void* ptr;
    gpuAssert(hipMallocManaged(&ptr, size));
    return ptr;
}

void dealloc(void* ptr, size_t size) {
    gpuAssert(hipFree(ptr));
}

#endif
