#include "hip/hip_runtime.h"
#ifndef SYNTH_GPU_H
#define SYNTH_GPU_H

#include <cstdint>
#include <thrust/scan.h>

#include "bitset_gpu.cu"
#include "expr.hpp"
#include "spec.hpp"
#include "synth.hpp"

#define BLOCK_SIZE 1024

struct DevicePassState {
    // Number of terms in the bank.
    uint32_t num_terms;

    // Did we find a solution in this pass?
    uint32_t found_sol;

    // If so, what's its index in the bank?
    uint32_t sol_idx;

    DevicePassState(uint32_t num_terms) :
            num_terms(num_terms),
            found_sol(false),
            sol_idx(0) {}
};

// See inclusive_scan.py for details.
__device__ inline void inclusive_scan(uint32_t* values, uint32_t length) {
    uint32_t step, idx;
    for (step = 1, idx = threadIdx.x * 2 + 1;
            step < length;
            step *= 2, __syncthreads()) {
        if (idx < length) {
            values[idx] += values[idx - step];
            idx = idx * 2 + 1;
        }
    }
    for (step = length / 4, idx = threadIdx.x * (2 * step) + (3 * step) - 1;
            step > 0;
            step /= 2, idx /= 2, __syncthreads()) {
        if (idx < length) {
            values[idx] += values[idx - step];
        }
    }
}

__global__ void pass_variable(
    int32_t height,
    DevicePassState* __restrict__ state,
    uint32_t result_mask,
    GPUBitset __restrict__ seen,
    uint32_t sol_result,
    uint32_t* __restrict__ term_results,
    uint32_t* __restrict__ term_lefts,
    uint32_t num_vars,
    const uint32_t* __restrict__ var_values,
    const int32_t* __restrict__ var_heights
) {
    uint32_t var_idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    uint32_t is_new = 0;
    uint32_t var_value;
    if (var_idx < num_vars && var_heights[var_idx] == height) {
        var_value = result_mask & var_values[var_idx];
        if (!GPUBitset_test_and_set(seen, var_value)) {
            is_new = 1;
        }
    }

    __shared__ uint32_t new_count[BLOCK_SIZE];
    new_count[threadIdx.x] = is_new;
    __syncthreads();
    inclusive_scan(new_count, BLOCK_SIZE);

    __shared__ uint32_t bank_segment_start;
    if (threadIdx.x == 0) {
        bank_segment_start = atomicAdd(&state->num_terms, new_count[BLOCK_SIZE - 1]);
    }
    __syncthreads();

    if (is_new) {
        uint32_t bank_idx = bank_segment_start + new_count[threadIdx.x] - 1;
        term_results[bank_idx] = var_value;
        term_lefts[bank_idx] = var_idx;
        if (var_value == sol_result) {
            state->found_sol = true;
            state->sol_idx = bank_idx;
        }
    }
}

class Synthesizer : public AbstractSynthesizer {
private:
    GPUBitset seen;
    DevicePassState* device_pass_state;

public:
    Synthesizer(Spec spec) : AbstractSynthesizer(spec),
            seen(GPUBitset_new(max_distinct_terms)) {
        DevicePassState state(num_terms);
        gpuAssert(hipMalloc(&device_pass_state, sizeof(DevicePassState)));
        gpuAssert(hipMemcpy(device_pass_state, &state, sizeof(DevicePassState),
                hipMemcpyHostToDevice));
    }

private:
    int64_t finish_pass() {
        hipDeviceSynchronize();

        DevicePassState state(0);
        gpuAssert(hipMemcpy(&state, device_pass_state, sizeof(DevicePassState),
                hipMemcpyDeviceToHost));
        num_terms = state.num_terms;
        return state.found_sol ? state.sol_idx : -1LL;
    }

    int64_t pass_Variable(int32_t height) {
        size_t vars_size = spec.num_vars * sizeof(uint32_t);
        uint32_t* device_var_values;
        int32_t* device_var_heights;
        gpuAssert(hipMalloc(&device_var_values, vars_size));
        gpuAssert(hipMalloc(&device_var_heights, vars_size));
        gpuAssert(hipMemcpy(device_var_values, &spec.var_values[0], vars_size,
                hipMemcpyHostToDevice));
        gpuAssert(hipMemcpy(device_var_heights, &spec.var_heights[0], vars_size,
                hipMemcpyHostToDevice));

        dim3 dim_grid(CEIL_DIV(spec.num_vars, BLOCK_SIZE));
        dim3 dim_block(BLOCK_SIZE);
        pass_variable<<<dim_grid, dim_block>>>(
            height,
            device_pass_state,
            result_mask,
            seen,
            spec.sol_result,
            term_results,
            term_lefts,
            spec.num_vars,
            device_var_values,
            device_var_heights
        );
        return finish_pass();
    }

    int64_t pass_Not(int32_t height) {
        return NOT_FOUND;
    }

    int64_t pass_And(int32_t height) {
        return NOT_FOUND;
    }

    int64_t pass_Or(int32_t height) {
        return NOT_FOUND;
    }

    int64_t pass_Xor(int32_t height) {
        return NOT_FOUND;
    }
};

#endif
