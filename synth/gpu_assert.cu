#ifndef ASSERT_GPU_H
#define ASSERT_GPU_H

#include <cassert>
#include <iostream>

#define gpuAssert(X) \
    do {                                                    \
        hipError_t error = X;                              \
        if (error != hipSuccess) {                         \
            std::cerr << #X                                 \
                    << ": " << hipGetErrorName(error)      \
                    << ": " << hipGetErrorString(error)    \
                    << std::endl;                           \
            std::exit(1);                                   \
        }                                                   \
    } while (0)

#endif
